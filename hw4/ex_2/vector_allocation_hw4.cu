#include <stdio.h>
#include <sys/time.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#define DataType double
#define n_streams 4


hipStream_t streams[n_streams];

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= len) return;
  out[idx] = in1[idx] + in2[idx];
  __syncthreads();
  return;
}

//@@ Insert code to implement timer start
std::time_t timer_start(){
  return std::time(nullptr);
}

//@@ Insert code to implement timer stop
std::time_t stop_timer(std::time_t start_time){
  return std::difftime(start_time, std::time(nullptr));
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  std::time_t start_time;


  //for random floating point numbers
  int highest = 1e6;
  int upper = 10e3;

  //@@ Insert code below to read in inputLength from args
  if(argc>1){
    inputLength = std::stoi(argv[1]);
  }
  else{
    std::cout << "Please provide input length";
    return -1;
  }

  

  printf("The input length is %d\n", inputLength);
  int S_seg = 131072;

  //@@ Insert code below to allocate Host memory for input and output
  //saving error messages and the memory pointer
  hostInput1 = (double*)  std::malloc(sizeof(DataType) * inputLength);
  hostInput2 = (double*)  std::malloc(sizeof(DataType) * inputLength);
  hostOutput = (double*)  std::malloc(sizeof(DataType) * inputLength);
  resultRef  = (double*)  std::malloc(sizeof(DataType) * inputLength);


  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  std::srand(std::time(nullptr));
  for(int i = 0; i<inputLength; i++)
  {
    hostInput1[i] = upper * (double)(std::rand() % highest) / highest;
    hostInput2[i] = upper *(double)(std::rand() % highest) / highest;
    hostOutput[i] = hostInput1[i] + hostInput2[i];
  }
  
  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, sizeof(DataType)*inputLength);
  hipMalloc(&deviceInput2, sizeof(DataType)*inputLength);
  hipMalloc(&deviceOutput, sizeof(DataType)*inputLength);


  

  //@@ Initialize the 1D grid and block dimensions here
  int TPB = 64;
  int cursor = 0;
  int segment = S_seg;
  for(int i = 0; i<((inputLength/S_seg)+1); i++){
    cursor = std::min(i * S_seg, inputLength);
    segment = std::min(S_seg, inputLength - cursor);
    //@@ Insert code to below to Copy memory to the GPU here
    hipMemcpyAsync(deviceInput1+cursor, hostInput1+cursor,  sizeof(DataType)*segment, hipMemcpyHostToDevice, streams[i%n_streams]);
    hipMemcpyAsync(deviceInput2+cursor, hostInput2+cursor,  sizeof(DataType)*segment, hipMemcpyHostToDevice, streams[i%n_streams]);
    //@@ Launch the GPU Kernel here
    vecAdd<<<(S_seg + TPB - 1)/TPB, TPB, 0, streams[i%n_streams]>>>(deviceInput1+cursor, deviceInput2+cursor, deviceOutput+cursor, segment);
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpyAsync(resultRef+cursor, deviceOutput+cursor , sizeof(DataType)*segment, hipMemcpyDeviceToHost, streams[i%n_streams]);
  }
  



  //@@ Insert code below to compare the output with the reference
  for(int i = 0; i < inputLength; i++){
        
    if(resultRef[i] != hostOutput[i]){

        std::cout << "Wrong computation!";
        return -1;
    }

  }
  std::cout << "Correct computation!";


  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  //@@ Free the CPU memory here
  std::free(hostInput1);
  std::free(hostInput2);
  std::free(hostOutput);

  return 0;
}
